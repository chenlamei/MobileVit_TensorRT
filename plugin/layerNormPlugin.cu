#include "hip/hip_runtime.h"
#include "layerNormPlugin.h"
#include <numeric>
using namespace nvinfer1;

PluginFieldCollection LayerNormPluginCreator::fc_{};
std::vector<PluginField> LayerNormPluginCreator::attr_;

constexpr int kWarpSize = 32;
template <typename T>
__inline__ __device__ void WarpReduceSum(T &local_data) {
#pragma unroll
  for (int mask = kWarpSize / 2; mask > 0; mask /= 2) {
    local_data += __shfl_down_sync(0xffffffff, local_data, mask, kWarpSize);
  }
}
template <typename T>
__inline__ __device__ void BlockReduceMean(T &local_data, T *sum_shared,
                                           T *mean_result, int len) {
  float warp_sum;
  const int lid = threadIdx.x & 0x1f;
  const int wid = threadIdx.x >> 5;
  WarpReduceSum(local_data);
  __syncthreads();
  if (lid == 0) {
    sum_shared[wid] = local_data;
  }
  __syncthreads();

  if (wid == 0) {
    if (threadIdx.x < blockDim.x / kWarpSize) {
      warp_sum = sum_shared[lid];
    } else {
      warp_sum = 0.0f;
    }
  }
  __syncwarp();
  WarpReduceSum(warp_sum);

  if (threadIdx.x == 0) {
    *mean_result = warp_sum / len;
  }
  __syncthreads();
}
template <typename T>
__global__ void LayerNormKernel(const T *__restrict__ pInput,
                                const T *__restrict__ gamma,
                                const T *__restrict__ beta,
                                T *__restrict__ pOutput, const int ld) {
  const int tx = threadIdx.x, index = blockIdx.x * ld + threadIdx.x;
  __shared__ T sum_shared[kWarpSize];
  __shared__ T mean_result;
  __shared__ T var_result;
  T value = 0, value_bak = 0;
  if (tx < ld) {
    value = __ldg(&pInput[index]);
    value_bak = value;
  }
  __syncwarp();
  BlockReduceMean(value, sum_shared, &mean_result, ld);
  if (tx < ld) {
    value = (value_bak - mean_result) * (value_bak - mean_result);
  }
  __syncwarp();
  BlockReduceMean(value, sum_shared, &var_result, ld);
  if (tx < ld) {
    pOutput[index] = (T)((float)(value_bak - mean_result) *
                             rsqrtf(float(var_result) + 1e-5f)) *gamma[tx] +
                         beta[tx];
  }
}

template <typename T>
__global__ void LayerNormNaiveKernel(const T *__restrict__ pInput,
                                     const T *__restrict__ gamma,
                                     const T *__restrict__ beta,
                                     T *__restrict__ pOutput, const int ld) {
  const int tx = threadIdx.x, index = blockIdx.x * ld + threadIdx.x;
  __shared__ float temp[256];
  float value = 0;
  if (tx < ld) {
    value = (float)pInput[index];
    temp[tx] = value;
  } else {
    temp[tx] = 0;
  }
  __syncthreads();

  for (int stride = 128; stride >= 1; stride /= 2) {
    if (tx < stride) {
      temp[tx] += temp[tx + stride];
    }
    __syncthreads();
  }
  float mean = temp[0] / ld;
  __syncthreads();
  if (tx < ld) {
    temp[tx] = (value - mean) * (value - mean);
  } else {
    temp[tx] = 0;
  }
  __syncthreads();

  for (int stride = 128; stride >= 1; stride /= 2) {
    if (tx < stride) {
      temp[tx] += temp[tx + stride];
    }
    __syncthreads();
  }
  float var = temp[0] / ld;
  // if(threadIdx.x==0){ printf("block %d mean:%f var:%f
  // stride:%d\n",blockIdx.x,mean,var,blockDim.x/2);}
  if (tx < ld) {
    pOutput[index] =
        (T)((value - mean) * rsqrtf(var + 1e-5f) * (float)gamma[tx] +
            (float)beta[tx]);
  }
}
int64_t volume(nvinfer1::Dims const& d)
{
    return std::accumulate(d.d, d.d + d.nbDims, 1, std::multiplies<int64_t>());
}
int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc,
                                 const PluginTensorDesc *outputDesc,
                                 const void *const *inputs,
                                 void *const *outputs, void *workspace,
                                 hipStream_t stream) noexcept {
                                 
  int bolck_num =volume(inputDesc[0].dims)/n_;
  
  const int blocksize = ((n_ - 1) / 32 + 1) * 32;

  // ld should be 144 or 192 or 240
  if (inputDesc[0].type == DataType::kFLOAT) {
    LayerNormKernel<<<bolck_num, blocksize, 0, stream>>>(
        (float *)inputs[0], weight_gpu_, bias_gpu_, (float *)outputs[0], n_);
    // LayerNormNaiveKernel <<<bolck_num, 256, 0, stream>>>((float
    // *)inputs[0],weight_gpu_,bias_gpu_, (float *)outputs[0],ld);
  } else if (inputDesc[0].type == DataType::kHALF) {
    LayerNormKernel<<<bolck_num, blocksize, 0, stream>>>(
        (__half *)inputs[0], weight_half_gpu_, bias_half_gpu_,
        (__half *)outputs[0], n_);
    // LayerNormNaiveKernel <<<bolck_num,256 , 0, stream>>>((__half
    // *)inputs[0],weight_half_gpu_,bias_half_gpu_, (__half *)outputs[0],ld);
  } else {
    printf("Unsupport datatype!\n");
  }
  return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);
